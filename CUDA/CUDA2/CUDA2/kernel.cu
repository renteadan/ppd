#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/*
 * This sample demonstrates how use texture fetches in CUDA
 *
 * This sample takes an input PGM image (image_filename) and generates
 * an output PGM image (image_filename_out).  This CUDA kernel performs
 * a simple 2D transform (rotation) on the texture coordinates (u,v).
 */

// Includes, system
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#ifdef _WIN32
#define WINDOWS_LEAN_AND_MEAN
#define NOMINMAX
#include <windows.h>
#endif

// Includes CUDA
#include <hip/hip_runtime.h>

// Utilities and timing functions
#include "helpers/helper_functions.h"  // includes hip/hip_runtime.h and hip/hip_runtime_api.h

// CUDA helper functions
#include "helpers/hip/hip_runtime_api.h"  // helper functions for CUDA error check

// Define the files that are to be save and the reference images for validation
// const char *imageFilename = "lena_bw.pgm";
const char *imageFilename = "test.pgm";
// const char *refFilename = "ref_rotated.pgm";

const char *sampleName = "simpleTexture";

////////////////////////////////////////////////////////////////////////////////
// Constants

float tx = 0.5f, ty = 0.5f;  // image translation
float scale = 1.0f;          // image scale
float cx, cy;                // image centre

// Auto-Verification Code
bool testResult = true;

////////////////////////////////////////////////////////////////////////////////
//! Transform an image using texture lookups
//! @param outputData  output data in global memory
////////////////////////////////////////////////////////////////////////////////
__device__ __host__ float lerp(float a, float b, float t) {
  return a + t * (b - a);
}

// higher-precision 2D bilinear lookup  // texture data type, return type
__device__ float tex2DBilinear(const hipTextureObject_t tex, float x,
                               float y) {
  x -= 0.5f;
  y -= 0.5f;
  float px = floorf(x);  // integer position
  float py = floorf(y);
  float fx = x - px;  // fractional position
  float fy = y - py;
  px += 0.5f;
  py += 0.5f;

  float ix1 =
      lerp(tex2D<float>(tex, px, py), tex2D<float>(tex, px + 1.0f, py), fx);
  float ix2 = lerp(tex2D<float>(tex, px, py + 1.0f),
                   tex2D<float>(tex, px + 1.0f, py + 1.0f), fx);

  float res = lerp(ix1, ix2, fy);
  return res;
}

__global__ void d_render(unsigned char *d_output, int width, int height,
                         float tx, float ty, float scale, float cx, float cy,
                         hipTextureObject_t texObj) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int i = y * width + x;

  float u = (x - cx) * scale + cx + tx;
  float v = (y - cy) * scale + cy + ty;

  if ((x < width) && (y < height)) {
    // write output color
    float c = tex2DBilinear(texObj, u, v);
    d_output[i] = c * 0xff;
  }
}

////////////////////////////////////////////////////////////////////////////////
// Declaration, forward
void runTest(int argc, char **argv);

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) {
  printf("%s starting...\n", sampleName);

  // Process command-line arguments
  if (argc > 1) {
    if (checkCmdLineFlag(argc, (const char **)argv, "input")) {
      getCmdLineArgumentString(argc, (const char **)argv, "input",
                               (char **)&imageFilename);
    }
  }

  runTest(argc, argv);

  printf("%s completed, returned %s\n", sampleName,
         testResult ? "OK" : "ERROR!");
  exit(0);
}

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void runTest(int argc, char **argv) {
  int devID = findCudaDevice(argc, (const char **)argv);

  // load image from disk
  unsigned char *hData = NULL;
  unsigned int width, height;
  char *imagePath = sdkFindFilePath(imageFilename, argv[0]);

  if (imagePath == NULL) {
    printf("Unable to source image file: %s\n", imageFilename);
    exit(EXIT_FAILURE);
  }

  sdkLoadPGM<unsigned char>(imagePath, &hData, &width, &height);

  unsigned int size = width * height * sizeof(unsigned char);
  printf("Loaded '%s', %d x %d pixels\n", imageFilename, width, height);

  // Allocate device memory for result
  unsigned char *dData = NULL;
  checkCudaErrors(hipMalloc((void **)&dData, size));

  // Allocate array and copy image data
  hipChannelFormatDesc channelDesc =
      hipCreateChannelDesc(8, 0, 0, 0, hipChannelFormatKindUnsigned);
  hipArray *cuArray;
  checkCudaErrors(hipMallocArray(&cuArray, &channelDesc, width, height));
  checkCudaErrors(hipMemcpy2DToArray(
      cuArray, 0, 0, hData, width * sizeof(unsigned char),
      width * sizeof(unsigned char), height, hipMemcpyHostToDevice));

  hipTextureObject_t tex;
  hipResourceDesc texRes;
  memset(&texRes, 0, sizeof(hipResourceDesc));

  texRes.resType = hipResourceTypeArray;
  texRes.res.array.array = cuArray;

  hipTextureDesc texDescr;
  memset(&texDescr, 0, sizeof(hipTextureDesc));

  texDescr.normalizedCoords = false;
  texDescr.filterMode = hipFilterModeLinear;
  texDescr.addressMode[0] = hipAddressModeWrap;
  texDescr.addressMode[1] = hipAddressModeWrap;
  texDescr.readMode = hipReadModeNormalizedFloat;

  checkCudaErrors(hipCreateTextureObject(&tex, &texRes, &texDescr, NULL));

  dim3 threadsPerBlock(32, 32, 1);
  dim3 numBlocks((width / threadsPerBlock.x) + 1,
                 (height / threadsPerBlock.y) + 1, 1);

  float tempx, tempy, sc;
  sc = atof(argv[1]);
  tempx = atof(argv[2]);
  tempy = atof(argv[3]);
  cx = width * tempx;
  cy = height * tempy;
  scale /= sc;

  checkCudaErrors(hipDeviceSynchronize());
  StopWatchInterface *timer = NULL;
  sdkCreateTimer(&timer);
  sdkStartTimer(&timer);

  // Execute the kernel
  d_render<<<numBlocks, threadsPerBlock>>>(dData, width, height, tx, ty, scale,cx, cy, tex);

  // Check if kernel execution generated an error
  getLastCudaError("Kernel execution failed");

  checkCudaErrors(hipDeviceSynchronize());
  sdkStopTimer(&timer);
  printf("Processing time: %f (ms)\n", sdkGetTimerValue(&timer));
  printf("%.2f Mpixels/sec\n",
         (width * height / (sdkGetTimerValue(&timer) / 1000.0f)) / 1e6);
  sdkDeleteTimer(&timer);

  // Allocate mem for the result on host side
  unsigned char *hOutputData = (unsigned char *)malloc(size);
  // copy result from device to host
  checkCudaErrors(hipMemcpy(hOutputData, dData, size, hipMemcpyDeviceToHost));

  // Write result to file
  char outputFilename[1024];
  strcpy(outputFilename, imagePath);
  strcpy(outputFilename + strlen(imagePath) - 4, "_out.pgm");
  sdkSavePGM(outputFilename, hOutputData, width, height);
  printf("Wrote '%s'\n", outputFilename);

  checkCudaErrors(hipDestroyTextureObject(tex));
  checkCudaErrors(hipFree(dData));
  checkCudaErrors(hipFreeArray(cuArray));
  free(imagePath);
}